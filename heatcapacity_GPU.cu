
#include <hip/hip_runtime.h>
/********************************************************************************
*
*   Copyright (C) 2015 Culham Centre for Fusion Energy,
*   United Kingdom Atomic Energy Authority, Oxfordshire OX14 3DB, UK
*
*   Licensed under the Apache License, Version 2.0 (the "License");
*   you may not use this file except in compliance with the License.
*   You may obtain a copy of the License at
*
*       http://www.apache.org/licenses/LICENSE-2.0
*
*   Unless required by applicable law or agreed to in writing, software
*   distributed under the License is distributed on an "AS IS" BASIS,
*   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
*   See the License for the specific language governing permissions and
*   limitations under the License.
*
********************************************************************************
*
*   Program: SPILADY - A Spin-Lattice Dynamics Simulation Program
*   Version: 1.0
*   Date:    Aug 2015
*   Author:  Pui-Wai (Leo) MA
*   Contact: info@spilady.ccfe.ac.uk
*   Address: Culham Centre for Fusion Energy, OX14 3DB, United Kingdom
*
********************************************************************************/

#if defined GPU

#include "spilady.h"
#include "prototype_GPU.h"

#if defined eltemp

// a functional form for the heat capacity C = a*tanh(bT) per atom is chosen.
__device__ double Ce_d(double Te){

    double a = 3e0;
    double b = 2e-4/boltz; //K^-1 / (eV K^-1)
    double C = a*tanh(b*Te);
    return C;
}

__device__ double Te_to_Ee_d(double Te){

    double a = 3e0;
    double b = 2e-4/boltz; //K^-1 / (eV K^-1)
    double Ee = a/b*log(cosh(b*Te));
    return Ee;

}

__device__ double Ee_to_Te_d(double Ee){

    double a = 3e0;
    double b = 2e-4/boltz; //K^-1 / (eV K^-1)
    double Te = acosh(exp(b/a*Ee))/b;
    return Te;
}

#endif

#endif
